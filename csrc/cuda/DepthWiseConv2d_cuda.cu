#include "hip/hip_runtime.h"
// Copyright (c) Samson Wang. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>


// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__global__ void DepthWiseConv2dFForward(const T* bottom_data,
    const T* weight_data,
    const T* bias_data,
    const int channels, const int padding, const int height,
    const int width, const int kernel_size,
    const int out_height, const int out_width,
    T* top_data) {

    T sum = 0;
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z % channels;
    //int n = blockIdx.z / channels;
    int o_idx = blockIdx.x * (blockDim.x - kernel_size + 1) + threadIdx.x;
    int o_idy = blockIdx.y * (blockDim.y - kernel_size + 1) + threadIdx.y;
    T bias = 0;
    if (bias_data != NULL) {
        bias = bias_data[c];
    }

//    int i_off_x = threadIdx.x - padding;
//    int i_off_y = threadIdx.y - padding;

    __shared__ T tmp_shared[32][32];
    __shared__ T w_shared[32];

    if (o_idx - padding >= 0 && o_idx - padding < width && o_idy - padding >=0 && o_idy - padding < height) {
        tmp_shared[threadIdx.y][threadIdx.x] = bottom_data[blockIdx.z * width * height + (o_idy - padding) * width + o_idx - padding];
//        printf("tids %d, %d, oid %d, %d, padding %d, width %d, height %d, block %d, %d\n", tidx, tidy, o_idx, o_idy, padding, width, height, blockDim.x, blockDim.y);
    } else {
        tmp_shared[threadIdx.y][threadIdx.x] = 0;
    }
    if (threadIdx.x < kernel_size * kernel_size) {
        w_shared[threadIdx.x] = weight_data[c * kernel_size * kernel_size + threadIdx.x];
    }
    __syncthreads();
//    std::cout << tidx << " " << tidy << " " << " o " << o_idx << "  " << o_idy << " padding " << padding << " " << width << std::endl;
    if (o_idx >= 0 && o_idx < out_width && o_idy >=0 && o_idy < out_height && threadIdx.x < blockDim.x - kernel_size + 1 && threadIdx.y < blockDim.y - kernel_size + 1) {
        for (int i = 0; i < kernel_size; i++) {
            for (int j = 0; j < kernel_size; j++) {
                sum += tmp_shared[threadIdx.y + i][threadIdx.x + j] * w_shared[i * kernel_size + j];
            }
        }
        top_data[blockIdx.z * out_width * out_height + (o_idy ) * out_width + o_idx ] = sum + bias;
    }

}

at::Tensor DepthWiseConv2d_forward_cuda(const at::Tensor& input,
                                const at::Tensor& weight,
                                const at::Tensor& bias,
                                const int stride,
                                const int padding,
                                const int dilation,
                                const int groups) {
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");

  auto batch_size = input.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto kernel_size = weight.size(2);

  auto out_height = (height - kernel_size + 1 + padding * 2) / stride;
  auto out_width = (width - kernel_size + 1 + padding * 2) / stride;
  AT_ASSERTM(weight.size(0) == channels, "Weight input channel must be equal to Input channel");

  auto output = at::empty({batch_size, channels, out_height, out_width}, input.options());
  auto blocks_x = THCCeilDiv((long)out_width, 32L-kernel_size+1);
  auto blocks_y = THCCeilDiv((long)out_height, 32L-kernel_size+1);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(blocks_x, blocks_y, batch_size * channels);
  dim3 block(32, 32);

//  std::cout << "SHAPE dim x " << blocks_x << " dim y " << blocks_y << " nc " << batch_size * channels << std::endl;

//  std::cout << channels << " " << padding << " " << height << " " << width << " " << kernel_size << std::endl;
  //printf("blockdim %d, %d, %d, griddim %d, %d, %d \n", block.x, block.y, block.z, grid.x, grid.y, grid.z);

  //if (output.numel() == 0) {
  //  THCudaCheck(hipGetLastError());
  //  return output;
  //}
//niu
//AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "ROIPool_forward", [&] {
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2d_forward", [&] {
    DepthWiseConv2dFForward<scalar_t><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         output.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return output;
}


