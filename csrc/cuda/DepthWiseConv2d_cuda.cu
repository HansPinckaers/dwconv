#include "hip/hip_runtime.h"
// Copyright (c) Samson Wang. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

// For small input computation
template <typename T, int FixedKernelSize>
__global__ void DepthWiseConv2dSmallFForward(const T* bottom_data,
    const T* weight_data,
    const T* bias_data,
    const int channels, const int padding, const int height,
    const int width, const int in_kernel_size,
    const int out_height, const int out_width, const int batch_size,
    T* top_data) 
{
    const int in_num = height * width;
    const int out_num = out_height * out_width;
    const int pad_height = height + padding*2;
    const int pad_width = width + padding*2;
    const int pad_num = pad_height * pad_width;
    const int kernel_size = FixedKernelSize > 0 ? FixedKernelSize : in_kernel_size;
    const int kernel_num = kernel_size * kernel_size;
    const int thread_num = blockDim.x * blockDim.y;
    const int n_steps = blockDim.z * gridDim.z;
    const int out_num_total = n_steps * out_num;
    const int in_num_total = n_steps * in_num;

    const int tidz = threadIdx.z + blockDim.z * blockIdx.z;
    const int tidx = blockDim.x * threadIdx.y + threadIdx.x;
    __shared__ T w_shared[8*32];
    __shared__ T tmp_shared[8*16*16];

    // Initialize tmp shared for input data
    for (int off = threadIdx.z * thread_num + tidx; off < 8 * 256; off += thread_num) {
        tmp_shared[off] = T(0);
    }

        T bias = T(0);
//        if (bias_data != NULL) bias = bias_data[c];

    __syncthreads();
    const int bound = batch_size * channels;
    const int pidx = pad_width * (threadIdx.y + padding) + threadIdx.x + padding;
    const int opidx = pad_width * threadIdx.y + threadIdx.x;
    int tmp_p_off = threadIdx.z * pad_num;
    int tmp_w_off = threadIdx.z * kernel_num;
    int tmp_off = tidx + tidz * in_num;
    int tmp_out_off = threadIdx.y * out_width + threadIdx.x + tidz * out_num;
    int half_pad_off = pad_width * blockDim.y;
    int half_in_off = width * blockDim.y;
    int half_out_off = out_width * blockDim.y;
    for (int n_off = 0; n_off < bound; n_off += n_steps) {
        int n_z = n_off + tidz;
        int c = n_z % channels;
        int c_off = c * kernel_num;
        if (n_z < bound) {
        // Load kernels into shared memory
            for (int off = tidx; off < kernel_num; off += thread_num) {
                w_shared[tmp_w_off + off] = weight_data[c_off + off];
            }

        // Load input data input shared memory, pay attention to the padding.
            tmp_shared[tmp_p_off + pidx] = bottom_data[tmp_off];
            if ((threadIdx.y + blockDim.y < height)) {
                tmp_shared[tmp_p_off + pidx + half_pad_off] = bottom_data[tmp_off + half_in_off];
            }
        }

        __syncthreads();
        /*
        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            printf("nz %d, bound %d, n_steps %d, n_off %d, blockDim.z %d, gridDim.z %d\n", n_z, bound, n_steps, n_off, blockDim.z, gridDim.z);
        }

        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            for (int z = 0; z < channels; z++) {
            for (int i = 0; i < pad_height + kernel_size ; i++) {
                for (int j = 0; j < pad_width + kernel_size ; j++) {
                    printf("%f ", tmp_shared[z * pad_num + i*pad_width + j]);
                }
                printf("\n");
            }
                printf("%d \n", z);
            }
            for (int z = 0; z < channels; z++) {
                for (int i = 0; i < kernel_size; i++) {
                    for (int j = 0; j < kernel_size; j++) {
                        printf("%f ", w_shared[z * kernel_num + i * kernel_size + j]);
                    }
                    printf("\n");
                }
            }
        }
        */

        if (n_z < bound && threadIdx.x < out_width && threadIdx.y < out_height) {
            // To do the math
            T sum = T(0);
            T sum1 = T(0);
            int i_poff = tmp_p_off + opidx;
            #pragma unroll
            for (int i = 0; i < kernel_num; i+= kernel_size) {
                #pragma unroll
                for (int j = 0; j < kernel_size; j++) {
                    const T f = w_shared[i + tmp_w_off + j];
                    sum += tmp_shared[i_poff + j] * f;
                    if ((threadIdx.y + blockDim.y < out_height)) {
                        sum1 += tmp_shared[i_poff + j + half_pad_off] * f;
                    }
                }
                i_poff += pad_width;
            }
//            sum += bias;
            top_data[tmp_out_off] = sum;
            if ((threadIdx.y + blockDim.y < out_height)) {
                top_data[tmp_out_off + half_out_off] = sum1;
                //printf("top data %d, %d, %d, %d\n", threadIdx.x, threadIdx.y, tmp_out_off, half_out_off);
            }
        }
        tmp_off += in_num_total;
        tmp_out_off += out_num_total;
        __syncthreads();
    }
}

template <typename T>
__global__ void DepthWiseConv2dFForward(
    const T* bottom_data,
    const T* weight_data,
    const T* bias_data,
    const int channels, 
    const int padding,
    const int height,
    const int width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int output_size,
    T* top_data)
{
  // indices relative to top_data, which outputs were gonna calc
  int o_idx = blockIdx.x * (blockDim.x - kernel_size + 1) + threadIdx.x;
  int o_idy = blockIdx.y * (blockDim.y - kernel_size + 1) + threadIdx.y;

  // current channel we're working on
  int c = (blockIdx.z) % channels;  

  T bias = 0;
  if (bias_data != NULL) {
    bias = bias_data[c];
  }

  __shared__ T w_shared[32];
  int tidx = threadIdx.y * blockDim.x + threadIdx.x;
  if (tidx < kernel_size * kernel_size) {
    w_shared[tidx] = weight_data[c * kernel_size * kernel_size + tidx];
  }

  __syncthreads();

  // tmp_shared is input data shared over block?
  __shared__ T tmp_shared[32*32];
  for (int n_off = 0; n_off < output_size; n_off += gridDim.z) {
    if (blockIdx.z + n_off < output_size) {
      T sum = 0;

      // boundary checks
      if (o_idx - padding >= 0 && 
          o_idx - padding < width && 
          o_idy - padding >= 0 && 
          o_idy - padding < height) 
      {
        tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = 
          bottom_data[(blockIdx.z + n_off) * width * height + (o_idy - padding) * width + o_idx - padding];
        // printf("tids %d, %d, oid %d, %d, padding %d, width %d, height %d, block %d, %d\n", tidx, tidy, o_idx, o_idy, padding, width, height, blockDim.x, blockDim.y);
      } else {
        tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = 0;
      }

      __syncthreads();

      //    std::cout << tidx << " " << tidy << " " << " o " << o_idx << "  " << o_idy << " padding " << padding << " " << width << std::endl;
      if (o_idx >= 0 && 
          o_idx < out_width && 
          o_idy >=0 && 
          o_idy < out_height && 
          threadIdx.x < blockDim.x - kernel_size + 1 && 
          threadIdx.y < blockDim.y - kernel_size + 1) 
      {
        for (int i = 0; i < kernel_size; i++) {
          for (int j = 0; j < kernel_size; j++) {
            sum += tmp_shared[(threadIdx.y + i) * blockDim.x + threadIdx.x + j] * w_shared[i * kernel_size + j];
          }
        }

        top_data[(n_off + blockIdx.z) * out_width * out_height + (o_idy) * out_width + o_idx] = sum + bias;
      }
    } else {
      // printf("blockDim %d, %d, %d. gridDim %d, %d, %d os %d z %d off %d ch %d\n", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, output_size, blockIdx.z, n_off, channels);
    }
    __syncthreads();
  }
}

template <typename T>
__global__ void DepthWiseConv2dTransposedFForward(
    const T* bottom_data,
    const T* weight_data,
    const T* bias_data,
    const int channels,
    const int padding,
    const int height,
    const int width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int output_size,
    T* top_data) 
{
  // indices relative to top_data, which outputs were gonna calc
  int o_idx = blockIdx.x * (blockDim.x - kernel_size + 1) + threadIdx.x;
  int o_idy = blockIdx.y * (blockDim.y - kernel_size + 1) + threadIdx.y;

  // current channel we're working on
  int c = (blockIdx.z) % channels;  

  T bias = 0;
  if (bias_data != NULL) {
    bias = bias_data[c];
  }

  __shared__ T w_shared[32];
  int tidx = threadIdx.y * blockDim.x + threadIdx.x;
  if (tidx < kernel_size * kernel_size) {
    w_shared[tidx] = weight_data[c * kernel_size * kernel_size + tidx];
  }

  __syncthreads();

  // tmp_shared is placeholder for input data shared over block
  // w_shared is weight data shared over block
  __shared__ T tmp_shared[32*32];
  for (int n_off = 0; n_off < output_size; n_off += gridDim.z) {
    if (blockIdx.z + n_off < output_size) {
      T sum = 0;

      // boundary checks
      if (o_idx - padding >= 0 && 
          o_idx - padding < width && 
          o_idy - padding >= 0 && 
          o_idy - padding < height) 
      {
        tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = 
          bottom_data[(blockIdx.z + n_off) * width * height + (o_idy - padding) * width + o_idx - padding];
        // printf("tids %d, %d, oid %d, %d, padding %d, width %d, height %d, block %d, %d\n", tidx, tidy, o_idx, o_idy, padding, width, height, blockDim.x, blockDim.y);
      } else {
        tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = 0;
      }

      __syncthreads();

      //    std::cout << tidx << " " << tidy << " " << " o " << o_idx << "  " << o_idy << " padding " << padding << " " << width << std::endl;
      if (o_idx >= 0 && 
          o_idx < out_width && 
          o_idy >=0 && 
          o_idy < out_height && 
          threadIdx.x < blockDim.x - kernel_size + 1 && 
          threadIdx.y < blockDim.y - kernel_size + 1) 
      {
        for (int i = 0; i < kernel_size; i++) {
          for (int j = 0; j < kernel_size; j++) {
            current += tmp_shared[(threadIdx.y) * blockDim.x + threadIdx.x] * w_shared[i * kernel_size + j];
            top_data[(n_off + blockIdx.z) * out_width * out_height + (o_idy + i) * out_width + o_idx + j] += current 
            // + bias;
          }
        }
      }
    } else {
      // printf("blockDim %d, %d, %d. gridDim %d, %d, %d os %d z %d off %d ch %d\n", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, output_size, blockIdx.z, n_off, channels);
    }

    __syncthreads();
  }
}

/* template <typename T> */
/* __global__ void DepthWiseConv2dLargeFForward(const T* bottom_data, */
/*     const T* weight_data, */
/*     const T* bias_data, */
/*     const int channels, const int padding, const int height, */
/*     const int width, const int kernel_size, */
/*     const int out_height, const int out_width, const int output_size, */
/*     T* top_data) { */
/*   int tidx = threadIdx.y * blockDim.x + threadIdx.x; */
/*   int c = (blockIdx.z) % channels; */
/*   T bias = 0; */
/*   if (bias_data != NULL) { */
/*     bias = bias_data[c]; */
/*   } */


/*   __shared__ T w_shared[32*32]; */
/*   if (tidx < kernel_size * kernel_size) { */
/*     w_shared[tidx] = weight_data[c * kernel_size * kernel_size + tidx]; */
/*   } */
/*   __syncthreads(); */

/*   __shared__ T tmp_shared[32*32]; */
/*   for (int n_off = 0; n_off < output_size; n_off += gridDim.z) { */
/*     if (blockIdx.z + n_off < output_size) { */
/*       T sum = 0; */
/*       //int n = blockIdx.z / channels; */
/*       //    int i_off_x = threadIdx.x - padding; */
/*       //    int i_off_y = threadIdx.y - padding; */


/*       if (o_idx - padding >= 0 && o_idx - padding < width && o_idy - padding >=0 && o_idy - padding < height) { */
/*         tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = bottom_data[(blockIdx.z + n_off) * width * height + (o_idy - padding) * width + o_idx - padding]; */
/*         //        printf("tids %d, %d, oid %d, %d, padding %d, width %d, height %d, block %d, %d\n", tidx, tidy, o_idx, o_idy, padding, width, height, blockDim.x, blockDim.y); */
/*       } else { */
/*         tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = 0; */
/*       } */
/*       __syncthreads(); */
/*       //    std::cout << tidx << " " << tidy << " " << " o " << o_idx << "  " << o_idy << " padding " << padding << " " << width << std::endl; */
/*       if (o_idx >= 0 && o_idx < out_width && o_idy >=0 && o_idy < out_height && threadIdx.x < blockDim.x - kernel_size + 1 && threadIdx.y < blockDim.y - kernel_size + 1) { */
/*         for (int i = 0; i < kernel_size; i++) { */
/*           for (int j = 0; j < kernel_size; j++) { */
/*             sum += tmp_shared[(threadIdx.y + i) * blockDim.x + threadIdx.x + j] * w_shared[i * kernel_size + j]; */
/*           } */
/*         } */
/*         top_data[(n_off + blockIdx.z) * out_width * out_height + (o_idy ) * out_width + o_idx ] = sum + bias; */
/*       } */
/*     } else { */
/*       //    printf("blockDim %d, %d, %d. gridDim %d, %d, %d os %d z %d off %d ch %d\n", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, output_size, blockIdx.z, n_off, channels); */
/*     } */
/*     __syncthreads(); */
/*   } */
/* } */

at::Tensor DepthWiseConv2d_forward_cuda(const at::Tensor& input,
    const at::Tensor& weight,
    const at::Tensor& bias,
    const int stride,
    const int padding,
    const int dilation,
    const int groups) {

  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");

  auto batch_size = input.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto kernel_size = weight.size(2);

  auto out_height = (height - kernel_size + 1 + padding * 2) / stride;
  auto out_width = (width - kernel_size + 1 + padding * 2) / stride;
  AT_ASSERTM(weight.size(0) == channels, "Weight input channel must be equal to Input channel");

  auto output = at::empty({batch_size, channels, out_height, out_width}, input.options());
  auto blockdim = 32;
  if (out_width < kernel_size && out_width + kernel_size - 1 < 32) {
    blockdim = kernel_size;
  } else if (out_width + kernel_size - 1 < 32) {
    blockdim = out_width + kernel_size - 1;
  }
  auto blocks_x = THCCeilDiv((long)out_width, blockdim-kernel_size+1L);
  auto blocks_y = THCCeilDiv((long)out_height, blockdim-kernel_size+1L);

  auto output_size = batch_size * channels;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto znum = output_size;
  if (znum > 2048) {
    znum = std::max((2048 / channels) * channels, channels);
  }
if (width + 2*padding > 16 || height + 2 * padding> 16) {
  dim3 grid(blocks_x, blocks_y, znum);
  dim3 block(blockdim, blockdim);

//  std::cout << "SHAPE dim x " << blocks_x << " dim y " << blocks_y << " nc " << batch_size * channels << std::endl;

//  std::cout << channels << " " << padding << " " << height << " " << width << " " << kernel_size << std::endl;
  //printf("blockdim %d, %d, %d, griddim %d, %d, %d outputsize %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, output_size);

  //if (output.numel() == 0) {
  //  THCudaCheck(hipGetLastError());
  //  return output;
  //}
//niu
//  printf("blockdim %d, %d, %d, griddim %d, %d, %d outputsize %d, channels %d, width %d, height %d, padding %d, stride %d, bias %s, kernel_size %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, batch_size, channels, width, height, padding, stride, bias.size(0), kernel_size);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2d_forward", [&] {
    DepthWiseConv2dFForward<scalar_t><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         output_size,
         output.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
} else {
  auto dimy = THCCeilDiv((long)height, 2L);
  auto blocks_x = 1;
  auto blocks_y = 1;
 
  dim3 grid(blocks_x, blocks_y, THCCeilDiv((long)channels*batch_size, 64L));
  dim3 block(width, dimy, 8);
//  printf("Small blockdim %d, %d, %d, griddim %d, %d, %d outputsize %d, channels %d, width %d, height %d, padding %d, stride %d, bias %s, kernel_size %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, batch_size, channels, width, height, padding, stride, bias.size(0), kernel_size);
if (kernel_size == 3) {
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
    DepthWiseConv2dSmallFForward<scalar_t, 3><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         batch_size,
         output.data<scalar_t>());
  });
} else if (kernel_size == 5) {
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
    DepthWiseConv2dSmallFForward<scalar_t, 5><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         batch_size,
         output.data<scalar_t>());
  });
} else {
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
    DepthWiseConv2dSmallFForward<scalar_t, 0><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         batch_size,
         output.data<scalar_t>());
  });

}
  THCudaCheck(hipGetLastError());

}
  return output;
}

std::vector<at::Tensor> DepthWiseConv2d_backward_weight_cuda(const at::Tensor& grad,
    const at::Tensor& input,
    const at::Tensor& weight,
    const at::Tensor& bias,
    const int stride,
    const int padding,
    const int dilation,
    const int groups) {

  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");

  input = input.contiguous().view({1, input.size(0) * input.shape(1), input.shape(2), input.shape(3)}) 
  grad = grad.contiguous().repeat(1, in_channels / groups, 1, 1)
  grad = grad.contiguous().view({grad_output.size(0) * grad_output.size(1), 1, grad_output.size(2), grad_output.size(3)})

  // ---------------------------------------- GRAD WEIGHT
  // First grad weight
  // the gradients of the weights can be calculated by 
  // convolving the output-gradient over the input
  auto batch_size = input.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto kernel_size = grad.size(2);  // kernel is the grad_output

  // TODO: this function does not yet support dilation
  auto out_height = (height - kernel_size + 1 + padding * 2) / stride;
  auto out_width = (width - kernel_size + 1 + padding * 2) / stride;
  AT_ASSERTM(grad.size(0) == channels, "Number of channels in the gradient must be equal to input channels");

  auto grad_weight_output = at::empty({batch_size, channels, out_height, out_width}, input.options());
  auto grad_weight = at::empty({channels, 1, kernel_size, kernel_size}, weight.options());

  auto blockdim = 32;
  if (out_width < kernel_size && out_width + kernel_size - 1 < 32) {
    blockdim = kernel_size;
  } else if (out_width + kernel_size - 1 < 32) {
    blockdim = out_width + kernel_size - 1;
  }
  auto blocks_x = THCCeilDiv((long)out_width, blockdim - kernel_size + 1L);
  auto blocks_y = THCCeilDiv((long)out_height, blockdim - kernel_size + 1L);

  auto output_size = batch_size * channels;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto znum = output_size;
  if (znum > 2048) {
    znum = std::max((2048 / channels) * channels, channels);
  }
  if (width + 2*padding > 16 || height + 2 * padding> 16) {
    dim3 grid(blocks_x, blocks_y, znum);
    dim3 block(blockdim, blockdim);
  
    //if (output.numel() == 0) {
    //  THCudaCheck(hipGetLastError());
    //  return output;
    //}
    //niu
    //  printf("blockdim %d, %d, %d, griddim %d, %d, %d outputsize %d, channels %d, width %d, height %d, padding %d, stride %d, bias %s, kernel_size %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, batch_size, channels, width, height, padding, stride, bias.size(0), kernel_size);
  
    AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2d_backward_weight", [&] {
      DepthWiseConv2dFForward<scalar_t><<<grid, block, 0, stream>>>(
           input.contiguous().data<scalar_t>(),
           grad.contiguous().data<scalar_t>(),
           bias.contiguous().data<scalar_t>(),
           channels,
           padding,
           height,
           width,
           kernel_size,
           out_height,
           out_width,
           output_size,
           grad_weight_output.data<scalar_t>());
    });
    THCudaCheck(hipGetLastError());
  }
}

std::vector<at::Tensor> DepthWiseConv2d_backward_input_cuda(const at::Tensor& grad,
    const at::Tensor& input,
    const at::Tensor& weight,
    const at::Tensor& bias,
    const int stride,
    const int padding,
    const int dilation,
    const int groups) {

  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");

  auto batch_size = input.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  if input_size is None:
      raise ValueError("grad.conv2d_input requires specifying an input_size")

  grad_input_padding = _grad_input_padding(grad_output, input_size, stride,
                                           padding, kernel_size)

  return torch.conv_transpose2d(
      grad_output, weight, None, stride, padding, grad_input_padding, groups,
      dilation)

  // ------------------------
  auto kernel_size = weight.size(2);  // kernel is the grad_output

  // TODO: this function does not yet support dilation
  auto out_height = (height - kernel_size + 1 + padding * 2) / stride;
  auto out_width = (width - kernel_size + 1 + padding * 2) / stride;
  AT_ASSERTM(grad.size(0) == channels, "Number of channels in the gradient must be equal to input channels");

  auto blockdim = 32;
  if (out_width < kernel_size && out_width + kernel_size - 1 < 32) {
    blockdim = kernel_size;
  } else if (out_width + kernel_size - 1 < 32) {
    blockdim = out_width + kernel_size - 1;
  }
  auto blocks_x = THCCeilDiv((long)out_width, blockdim - kernel_size + 1L);
  auto blocks_y = THCCeilDiv((long)out_height, blockdim - kernel_size + 1L);

  auto grad_input = at::empty({batch_size, channels, height, width}, input.options());

  auto output_size = batch_size * channels;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto znum = output_size;
  if (znum > 2048) {
    znum = std::max((2048 / channels) * channels, channels);
  }
  if (width + 2*padding > 16 || height + 2 * padding> 16) {
    dim3 grid(blocks_x, blocks_y, znum);
    dim3 block(blockdim, blockdim);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2d_backward_input", [&] {
      DepthWiseConv2dFForward<scalar_t><<<grid, block, 0, stream>>>(
           input.contiguous().data<scalar_t>(),
           grad.contiguous().data<scalar_t>(),
           bias.contiguous().data<scalar_t>(),
           channels,
           padding,
           height,
           width,
           kernel_size,
           out_height,
           out_width,
           output_size,
           grad_weight_output.data<scalar_t>());
    });
    THCudaCheck(hipGetLastError());
  }
  return output;
}

